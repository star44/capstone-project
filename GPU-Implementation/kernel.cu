#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <cstdio>
#include <cstring>
#include <ctime>
#include <iostream>
#include <string>
#include <fstream>
#include <stdlib.h>
#include <vector>
#include <thread>

#include "file_handling.h"

#define NUM_ROWS 120000000
#define ARRAY_HEIGHT 1000
#define ARRAY_WIDTH 48
#define BLOCK_DIMS 2560
#define GRID_DIMS 256

#define NUM_THREADS 6
#define MATRIX_SIZE (ARRAY_WIDTH * ARRAY_HEIGHT)
#define CHUNK_STRING_SIZE (MAX_LINE_LENGTH * ARRAY_HEIGHT)
#define NUM_CHUNKS (NUM_ROWS / ARRAY_HEIGHT)

const char* PATH_TO_MAX_VALS = "../../max_values_by_user.csv";
const char* PATH_TO_NMI_DATA = "../../nmi_data_20181029.csv";

hipError_t normalise(char* chunk, char* dev_chunk, float* vals, float* dev_vals, int num_threads);

__global__ void divide(char* dev_chunk, float* dev_vals) {
	int row = threadIdx.x;
	char* ptr = dev_chunk;
	int max;
	
	// scroll the pointer to where it needs to be in the chunk (each thread gets a row from the chunk)
	for (int i = 0; i < row; i++) {
		while (*ptr != '\n') {
			ptr++;
		}
		ptr++; // scroll past the '\n' symbol
	}

	// scroll past first couple columns...
	while (*ptr != ',') {
		ptr++;
	}
	ptr++;
	while (*ptr != ',') {
		ptr++;
	}
	ptr++;
	// read each column into the dev_vals 
	for (int col = 0; col < ARRAY_WIDTH; col++) {
		dev_vals[row * ARRAY_WIDTH + col] = 0.0;
		int factor = 0;
		int decimals = 0;
		while (*ptr != '\n' && *ptr != ',') {
			if (decimals == 0) {
				if (*ptr == '.') {
					factor = -1;
					decimals = 1;
					continue;
				}
				dev_vals[row * ARRAY_WIDTH + col] = dev_vals[row * ARRAY_WIDTH + col] * pow(10.0, factor) + (*ptr - '0');
				factor++;
			}
			else {
				//dev_vals[row * ARRAY_WIDTH + col] = dev_vals[row * ARRAY_WIDTH + col] + (*ptr - '0') * pow(10.0, factor);
				factor--;
			}
			ptr++;
		}
		ptr++;
	}
}


int main() {

	hipError_t cudaStatus;

	std::FILE* dataset_fp = fopen(PATH_TO_NMI_DATA, "r");
	char* chunk = new char[ARRAY_HEIGHT];
	char* dev_chunk; float* dev_vals;
	char line[MAX_LINE_LENGTH];
	char* token;
	char* lineToken;

	int linesRead = roundedReadChunk(0.95 * ARRAY_HEIGHT, dataset_fp, chunk); // read 900MB into the chunk (plus whatever is needed to get to end of line)
	float* vals = new float[linesRead * ARRAY_WIDTH];
	cudaStatus = hipMalloc((void**) &dev_chunk, ARRAY_HEIGHT * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**) &dev_vals, linesRead * ARRAY_WIDTH * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
	std::fgets(chunk, MAX_LINE_LENGTH, dataset_fp);
	normalise(chunk, dev_chunk, vals, dev_vals, linesRead);
	for (int i = 0; i < linesRead * ARRAY_WIDTH; i++) {
		if (i % ARRAY_WIDTH == 0) {
			std::cout << std::endl;
		}
		std::cout << vals[i] << ", ";
	}

	delete [] chunk; // free host memory
	delete [] vals;
	hipFree(dev_chunk); // free gpu memory
	hipFree(dev_vals);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	return 0;
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t normalise(char* chunk, char* dev_chunk, float* vals, float* dev_vals, int num_threads) {
	
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	// Copy chunk from host to device
	cudaStatus = hipMemcpy(dev_chunk, chunk, ARRAY_HEIGHT * sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed when copying from host to device!\n");
	}

	// Launch a kernel on the GPU with one thread for each element.
	divide <<<1, num_threads>>> (dev_chunk, dev_vals);

	std::cout << "Finished computing from GPU" << std::endl;

	// Copy data from device to host
	cudaStatus = hipMemcpy(vals, dev_vals, num_threads * ARRAY_WIDTH * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed when copying from device to host!\n");
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	}
	return cudaStatus;
}